#include "hip/hip_runtime.h"
// LAB1b.cpp : This file contains the 'main' function. Program execution begins and ends there.
//

#include <iostream>
#include <fstream>
#include <string>

class Good{
public:
    std::string Name;
    int Amount;
    float Price;

    Good() {}

    Good(std::string parts[]){
        Name = parts[0];
        Amount = std::atoi(parts[1].c_str());
        Price = std::atoi(parts[2].c_str());
    }
    
    int sizeOfName(){
        return Name.size();
    }
};

class Goods{
public:
    Good Array[30];

    int size(){
        return sizeof(Array) / sizeof(Array[0]);
    }
};

Goods* readGoodsFromFile(std:: string fileName, std::string seperator){
    Goods* goods = new Goods();
    std::string line;
    std::ifstream stream(fileName);
    int currLine = 0;
    std::string parsedItems[3];
    if(stream.is_open()){
        while(std::getline(stream, line))
        {
            int currChunk = 0;
            int pos = 0;
            std::string part;
            while((pos = line.find(seperator)) != std::string::npos){
                part = line.substr(0,pos);
                parsedItems[currChunk++] = part;
                line.erase(0, pos + 1);
            }
            goods->Array[currLine++] = Good(parsedItems);
        }
        stream.close();
    }
    return goods;
}

int maxSize(Goods goods){
    int max = goods.Array[0].sizeOfName();
    for(int i = 1; i < goods.size(); i++){
        int temp = goods.Array[i].sizeOfName();
        if(temp > max)
            max = temp;
    }
    return max;
}

void copyItemsToArrays(Goods goods, char* names, int* namesLenght, int* amount, float* price){
    int curNamesPos = 0;
    for( int i = 0; i < goods.size(); i++){
        Good curGood = goods.Array[i];
        amount[i] = curGood.Amount;
        price[i] = curGood.Price;

        int len = std::string::strlen(curGood.Name);
        namesLenght[i] = len;
        for(int j = 0; i < len; j++){
            names[curNamesPos++] = curGood.Name[j];
        }
    }
}


int main(){
    int threadCount = 4;
    auto goods = readGoodsFromFile("IFF-8-1_PuzinasA_L1a_dat_1.txt", ";");
    int maxResultSize = maxSize(*goods) + 5;


    std::cout<<"HELLO";
}

