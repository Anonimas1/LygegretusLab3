// LAB1b.cpp : This file contains the 'main' function. Program execution begins and ends there.
//


#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <string.h>
#include <stdlib.h>

class Good{
public:
    std::string Name;
    int Amount;
    float Price;

    Good() {}

    Good(std::string parts[]){
        Name = parts[0];
        Amount = std::atoi(parts[1].c_str());
        Price = std::atoi(parts[2].c_str());
    }
    
    int sizeOfName(){
        return Name.size();
    }
};

class Goods{
public:
    Good Array[30];

    int size(){
        return sizeof(Array) / sizeof(Array[0]);
    }
};

Goods* readGoodsFromFile(std:: string fileName, std::string seperator){
    Goods* goods = new Goods();
    std::string line;
    std::ifstream stream(fileName);
    int currLine = 0;
    std::string parsedItems[3];
    if(stream.is_open()){
        while(std::getline(stream, line))
        {
            int currChunk = 0;
            int pos = 0;
            while((pos = line.find(seperator)) != std::string::npos){
                std::string part = line.substr(0,pos);
                parsedItems[currChunk++] = part;
                line.erase(0, pos + 1);
            }
            parsedItems[2] = line.substr(0, line.find('\n'));
            goods->Array[currLine++] = Good(parsedItems);
        }
        stream.close();
    }
    return goods;
}

int maxSize(Goods goods){
    int max = goods.Array[0].sizeOfName();
    for(int i = 1; i < goods.size(); i++){
        int temp = goods.Array[i].sizeOfName();
        if(temp > max)
            max = temp;
    }
    return max;
}

void copyItemsToArrays(Goods goods, char* names, int* namesLenght, int* nameChunkSize, int* amounts, float* prices){
    int curNamesPos = 0;
    int curChunk = 0;
    for( int i = 0; i < goods.size(); i++){
        Good curGood = goods.Array[i];
        amounts[i] = curGood.Amount;
        prices[i] = curGood.Price;

        int len = curGood.Name.length();
        namesLenght[i] = len;
        for(int j = 0; j < len; j++){ 
            names[curNamesPos++] = curGood.Name[j];
        }
        curChunk++;
        curNamesPos = curChunk * *nameChunkSize;
    }
}

void writeToFile(std::string fileName, char* results){
    std::ofstream stream(fileName);
    if(stream.is_open()){
        stream << results;
    }
    stream.close();
}

__global__ void proccesGoods(char* names, int* namesLenght, int* nameChunkSize, int* amounts, float* prices, int* arraySize, int* resultCount, char* results);
__device__ char* getName(char* names, int lenghtOfName, int index, int nameChunkSize);
__device__ char* getResult(char* name, int nameLenght, int amount, float price);
__device__ int fieldSum(int amount, float price);
__device__ char* addNumberToName(char* name, int nameLenght, int number);
__device__ void setSliceBoundaries(int* startIndex, int* endIndex, int arrSize);
__device__ void writeResult(char* results, char* result, int resultSize, int index, int maxResultSize);
__device__ bool isAceptable(char* data);
int main(){
    std::string resultFile ="IFF-8-1_PuzinasA_L3_rez.txt";
    //std::string dataFile = "IFF-8-1_PuzinasA_L1a_dat_1.txt";
    std::string dataFile = "IFF-8-1_PuzinasA_L1a_dat_2.txt";
    //std::string dataFile = "IFF-8-1_PuzinasA_L1a_dat_3.txt";
    int threadCount = 4;
    auto goods = readGoodsFromFile(dataFile, ";");
    int maxNameSize = maxSize(*goods) + 5;
    int sizeOfArray = goods->size();
    int resultCount = 0;

    size_t namesArrSize = sizeof(char) * maxNameSize * sizeOfArray;
    size_t namesLenghtArrSize = sizeof(int) * sizeOfArray;
    size_t amountsArrSize = sizeof(int) * sizeOfArray;
    size_t pricesArrSize = sizeof(float) * sizeOfArray;
    size_t resultArrSize = sizeof(char) * (maxNameSize + 5) * sizeOfArray;

    char* names = (char*)malloc(namesArrSize);
    int* namesLenght = (int*)malloc(namesLenghtArrSize);
    int* amounts = (int*)malloc(amountsArrSize);
    float* prices = (float*)malloc(pricesArrSize);    

    copyItemsToArrays(*goods, names, namesLenght, &maxNameSize, amounts, prices);  
      
    //Cuda name arrays
    char* cudaNames;
    int* cudaNamesLenght;
    int* cudaMaxNameSize;
    //Cuda number arrays
    int* cudaAmounts;
    float* cudaPrices;
    //Cuda array size
    int* cudaSizeOfArray;
    int* cudaResultCount;
    //Cuda result
    char* cudaResult;
    //Cuda memory allocation
    hipMalloc(&cudaNames, namesArrSize);
    hipMalloc(&cudaNamesLenght, namesLenghtArrSize);
    hipMalloc(&cudaMaxNameSize, sizeof(int));
    
    hipMalloc(&cudaAmounts, amountsArrSize);
    hipMalloc(&cudaPrices, pricesArrSize);
    
    hipMalloc(&cudaSizeOfArray, sizeof(int));
    hipMalloc(&cudaResultCount, sizeof(int));

    hipMalloc(&cudaResult, resultArrSize);
    //-------------------------------------
    //Cuda memory copy
    hipMemcpy(cudaNames, names, namesArrSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaNamesLenght, namesLenght, namesLenghtArrSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaMaxNameSize, &maxNameSize, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(cudaAmounts, amounts, amountsArrSize, hipMemcpyHostToDevice);
    hipMemcpy(cudaPrices, prices, pricesArrSize, hipMemcpyHostToDevice);

    hipMemcpy(cudaSizeOfArray, &sizeOfArray, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cudaResultCount, &resultCount, sizeof(int), hipMemcpyHostToDevice);
    //-----------------------------
    proccesGoods<<<1, threadCount>>>(
        cudaNames,
        cudaNamesLenght,
        cudaMaxNameSize,
        cudaAmounts,
        cudaPrices,
        cudaSizeOfArray,
        cudaResultCount,
        cudaResult
    );
    
    hipDeviceSynchronize();

    char* results = (char*)malloc(resultArrSize);
    hipMemcpy(results, cudaResult, resultArrSize, hipMemcpyDeviceToHost);
    writeToFile(resultFile, results);

}

__global__ void proccesGoods(char* names, int* namesLenght, int* nameChunkSize, int* amounts, float* prices, int* arraySize, int* resultCount, char* results){
	int startIndex = 0;
	int endIndex = 0;
    setSliceBoundaries(&startIndex, &endIndex, *arraySize);
    for(int i = startIndex; i < endIndex; i++){
        char* name = getName(names, namesLenght[i], i, *nameChunkSize);
        char* result = getResult(name, namesLenght[i], amounts[i], prices[i]);
        if(isAceptable(result)){
            int index = atomicAdd(resultCount, namesLenght[i] + 6);
            writeResult(results, result, namesLenght[i] + 5, index, *nameChunkSize + 5);
        }
    }

}
__device__ bool isAceptable(char* data){
    if(data[0] < 69)
        return true;

    return false;
}

__device__ void writeResult(char* results, char* result, int resultSize, int index, int maxResultSize){
    for(int i = index, j = 0; j < resultSize; i++, j++){
        results[i] = result[j];
    }
    results[index + resultSize] = '\n';
}


__device__ char* getName(char* names, int lenghtOfName, int index, int nameChunkSize){
    int start = index * nameChunkSize;
    char* result = new char[lenghtOfName];
    for(int i = 0; i < lenghtOfName; i++){
        result[i] = names[i + start];
    }
    return result;
}

__device__ char* getResult(char* name, int nameLenght, int amount, float price){
    int sum = fieldSum(amount, price);
    return addNumberToName(name, nameLenght, sum);
}

__device__ int fieldSum(int amount, float price){
    return (int)(amount + (int)price);
}
__device__ char* addNumberToName(char* name, int nameLenght, int number){
    char* result = new char[nameLenght + 5];
    for(int i = 0; i < nameLenght; i++){
        result[i] = name[i];
    }


    int currIndex = nameLenght;
    result[currIndex++] = '-';
    result[currIndex++] = (number / 1000) + '0';
    number = number - (number / 1000) * 1000;
    result[currIndex++] = (number / 100) + '0';
    number = number - (number / 100) * 100;
    result[currIndex++] = (number / 10) + '0';
    number = number - (number / 10) * 10;
    result[currIndex] = (number % 10) + '0';    
    return result;
}

__device__ void setSliceBoundaries(int* startIndex, int* endIndex, int arrSize){
    int chunkSize = arrSize / blockDim.x;
    *startIndex = chunkSize * threadIdx.x;
	*endIndex = (threadIdx.x == blockDim.x - 1) ? arrSize : chunkSize * (threadIdx.x + 1);
}

